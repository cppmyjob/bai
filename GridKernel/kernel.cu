#include <hip/hip_runtime.h>

extern "C"
__global__ void Grid(int* blockDimX, int* blockIdxX, int* threadIdxX)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    blockDimX[i] = blockDim.x;
    blockIdxX[i] = blockIdx.x;
    threadIdxX[i] = threadIdx.x;
}
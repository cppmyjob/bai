#include <hip/hip_runtime.h>

// Complex data type
typedef float2 Complex;

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
	Complex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
	Complex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
	Complex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// Complex pointwise multiplication
extern "C"
__global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}

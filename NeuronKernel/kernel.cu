#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C"
__global__ void Grid(const float* input, float* weights, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    blockDimX[i] = blockDim.x;
    blockIdxX[i] = blockIdx.x;
    threadIdxX[i] = threadIdx.x;
}